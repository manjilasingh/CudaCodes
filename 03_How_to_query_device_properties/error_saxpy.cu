#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Intentional out-of-bounds access
  y[i + n * 100] = a*x[i] + y[i];
}

int main(void)
{
  int N = 20 * (1 << 20);
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(start);

  // Invalid configuration - too many threads
  saxpy<<<(N+255)/256, 8256>>>(N, 2.0, d_x, d_y);
hipError_t errSync  = hipGetLastError();
hipError_t errAsync = hipDeviceSynchronize();
if (errSync != hipSuccess) 
  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
if (errAsync != hipSuccess)
  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

  
  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
  }

  printf("Max error: %f\n", maxError);
}